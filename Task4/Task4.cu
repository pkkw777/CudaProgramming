#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 100000000

float hArray[N];
float *dArray;
int blocks;
clock_t begin1,begin2,begin3,begin4,end1,end2,end3,end4;

void prologue(void)
{
        memset(hArray, 0, sizeof(hArray));
        for(int i = 0; i < N; i++)
        {
                hArray[i] = i + 1;
        }
        hipMalloc((void**)&dArray, sizeof(hArray));
        begin2 = clock();
        hipMemcpy(dArray, hArray, sizeof(hArray), hipMemcpyHostToDevice);
        end2 = clock();
}

void epilogue(void)
{

        hipMemcpy(hArray, dArray, sizeof(hArray), hipMemcpyDeviceToHost);
        hipFree(dArray);
}

// Kernel
__global__ void pow3(float *A)
{
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        if(x < N)
                {
                        #pragma unroll
                        for(int i =0; i< 100; i++)
                        {
                A[x] += A[x] * A[x] * A[x] + A[x] * A[x] + A[x];
                        }
                }
}

__global__ void pow4(float *A)
{
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        if(x < N)
                {
                        #pragma unroll 1
                        for(int i =0; i< 100; i++)
                        {
                A[x] += A[x] * A[x] * A[x] + A[x] * A[x] + A[x];
                        }
                }
}

//CPU
void cpu(float *A)
{
        int x;
        for (x = 0; x<N; x++)
        {
                A[x] = A[x] * A[x] * A[x] * A[x] * A[x] * A[x];
        }
}

int main(int argc, char** argv)
{
        int devCnt;
        hipGetDeviceCount(&devCnt);

        if(devCnt == 0) {
                perror("No CUDA devices available -- exiting.");
                return 1;
        }

        struct hipDeviceProp_t *prop;
        prop = (hipDeviceProp_t*)malloc(sizeof(struct hipDeviceProp_t));
        hipGetDeviceProperties(prop,0);
        printf("Ilosc watkow: %d\n", prop->maxThreadsPerBlock);

        //GPU
        prologue();
        blocks = N / prop->maxThreadsPerBlock;
        if(N % prop->maxThreadsPerBlock)
                blocks++;

                begin4 = clock();
        pow3<<<blocks, prop->maxThreadsPerBlock>>>(dArray);
                hipDeviceSynchronize();
                end4 = clock();

                begin1 = clock();
                pow4<<<blocks, prop->maxThreadsPerBlock>>>(dArray);
                hipDeviceSynchronize();
                end1 = clock();

        epilogue();

        //CPU
        begin3 = clock();
        cpu(hArray);
        end3 = clock();

        double time_spent1 = (double)(end1 - begin1) / CLOCKS_PER_SEC;

        double time_spent4 = (double)(end4 - begin4) / CLOCKS_PER_SEC;

        printf("Nie odwiniete: %f\n", time_spent1);
        printf("Odwiniete: %f\n", time_spent4);

        return 0;
}


